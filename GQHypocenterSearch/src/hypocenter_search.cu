#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <jni.h>

#include "travel_table.hpp"
#include "globalquake.hpp"
#include "geo_utils.hpp"
#include "globalquake_jni_GQNativeFunctions.h"

#define BLOCK 128
#define PHI 1.61803398875
#define DEPTH_RESOLUTION 1.0

#define STATION_FILEDS 4

float* travel_table_device;

__device__ void moveOnGlobe(float fromLat, float fromLon, float angle, float distance, float* lat, float* lon)
{
    // calculate angles
    float delta = distance / EARTH_RADIUS;
    float theta = fromLat;
    float phi = fromLon;
    float gamma = angle;

    // calculate sines and cosines
    float c_theta = cosf(theta);
    float s_theta = sinf(theta);
    float c_phi = cosf(phi);
    float s_phi = sinf(phi);
    float c_delta = cosf(delta);
    float s_delta = sinf(delta);
    float c_gamma = cosf(gamma);
    float s_gamma = sinf(gamma);

    // calculate end vector
    float x = c_delta * c_theta * c_phi - s_delta * (s_theta * c_phi * c_gamma + s_phi * s_gamma);
    float y = c_delta * c_theta * s_phi - s_delta * (s_theta * s_phi * c_gamma - c_phi * s_gamma);
    float z = s_delta * c_theta * c_gamma + c_delta * s_theta;

    // calculate end lat long
    *lat = asinf(z);
    *lon = atan2f(y, x);
}

__device__ void calculateParams(int points, int index, float maxDist, float fromLat, float fromLon, float* lat, float* lon, float* dist) {
    float ang = 2 * M_PI / (PHI * PHI) * index;
    *dist = sqrtf(index) * (maxDist / sqrtf(points));
    moveOnGlobe(fromLat, fromLon, ang, *dist, lat, lon);
}

__global__ void evaluateHypocenter(size_t points, float maxDist, float fromLat, float fromLon, float max_depth)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    float depth = max_depth * (blockIdx.y / (float)blockDim.y); 
    float lat, lon, dist;
    calculateParams(points, index, maxDist, fromLat, fromLon, &lat, &lon, &dist);
}

bool run_hypocenter_search(float* stations, size_t station_count, size_t points, float maxDist, float fromLat, float fromLon)
{
    bool success = true;
    size_t station_array_size = sizeof(float) * station_count * STATION_FILEDS;
    float* d_stations;
    success &= hipMalloc(&d_stations, station_array_size) == hipSuccess;
    success &= hipMemcpy(d_stations, stations, station_array_size, hipMemcpyHostToDevice) == hipSuccess;

    dim3 blocks = {(unsigned int)ceil(points / BLOCK), (unsigned int)ceil(max_depth / DEPTH_RESOLUTION), 1};
    dim3 threads = {BLOCK, 1, 1};
    
    printf("%d %d %d\n", blocks.x, blocks.y, blocks.z);
    printf("%d %d %d\n", threads.x, threads.y, threads.z);
    printf("total points: %lld\n", (((long long)(blocks.x * blocks.y * blocks.z)) * (long long)(threads.x * threads.y * threads.z)));

    if(success) evaluateHypocenter<<<blocks, threads>>>(points, maxDist, fromLat, fromLon, max_depth);
    success &= hipDeviceSynchronize();
    hipError_t err = hipGetLastError();

    if(d_stations) hipFree(d_stations);

    return success;
}


JNIEXPORT jfloatArray JNICALL Java_globalquake_jni_GQNativeFunctions_findHypocenter
  (JNIEnv *env, jclass, jobjectArray stations, jfloat fromLat, jfloat fromLon, jlong points, jfloat maxDist){
    size_t station_count = env->GetArrayLength(stations);
    
    float* stationsArray = static_cast<float*>(malloc(sizeof(float) * station_count * STATION_FILEDS));
    if(!stationsArray){
        goto cleanup;
    }

    for(int i = 0; i < station_count; i++){
        jfloatArray oneDim = (jfloatArray)env->GetObjectArrayElement(stations, i);
        jfloat *element = env->GetFloatArrayElements(oneDim, 0);
        
        for(int j = 0; j < STATION_FILEDS; j++){
            stationsArray[i * STATION_FILEDS + j] = element[j];    
        }
    }

    run_hypocenter_search(stationsArray, station_count, points, maxDist, fromLat, fromLon);

    if(stationsArray) free(stationsArray);

    cleanup:

    for (int i = 0; i < station_count; i++) {
        jfloatArray oneDim = (jfloatArray) env->GetObjectArrayElement(stations, i);
        jfloat *elements = env->GetFloatArrayElements(oneDim, 0);

        env->ReleaseFloatArrayElements(oneDim, elements, 0);
        env->DeleteLocalRef(oneDim);
    }

    return nullptr;
}

/*
 * Class:     globalquake_jni_GQNativeFunctions
 * Method:    initCUDA
 * Signature: ()Z
 */
JNIEXPORT jboolean JNICALL Java_globalquake_jni_GQNativeFunctions_initCUDA
      (JNIEnv *, jclass){
    size_t table_size = sizeof(float) * table_columns * table_rows;
    bool success = true;
    success &= hipMalloc(&travel_table_device, table_size) == hipSuccess;
    success &= hipMemcpy(travel_table_device, p_wave_table, table_size, hipMemcpyHostToDevice) == hipSuccess;
    return success;
}
